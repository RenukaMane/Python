#include "hip/hip_runtime.h"
#include <stdio.h>

//cuda headers
#include <hip/hip_runtime.h>
#include "helper_timer.h"

//macros
#define BLOCK_WIDTH 1032

//global variables
int *hostA = NULL;
int *hostB = NULL;
int *hostC = NULL;
int *gold = NULL;

int *deviceA = NULL;
int *deviceB = NULL;
int *deviceC = NULL;

float timeOnCPU = 0.0f;
float timeOnGPU = 0.0f;

// cuda kernel function
__global__ void matMulGPU(int *A, int *B, int *C, int numARows, int numAColumns, int numBColumns, int numCColumns)
{
    //variable declarations
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int column = blockIdx.x * blockDim.x + threadIdx.x;

    // code
    if((row < numARows) && (column < numBColumns))
    {
        int value = 0.0;
        
        for(int k = 0; k < numAColumns; k++)
        {
            int a = A[row * numAColumns + k];
            int b = B[k * numBColumns + column];
            value += a * b;
        }

        C[row * numCColumns + column] = value;
    }

}

int main()
{
    // function declarations
    void InitA(int *data,int,int);
    void InitB(int *data,int,int);
    void matMulCPU(int *, int *, int *, int, int, int, int);
    void cleanup(void);

    // variable declaration
    int numARows = BLOCK_WIDTH;
    int numAColumns = BLOCK_WIDTH;
    int numBRows = BLOCK_WIDTH;
    int numBColumns = BLOCK_WIDTH;
    int numCRows = numARows;
    int numCColumns = numBColumns;

    int numGoldRows = numARows;
    int numGodlColumns = numBColumns;

    int sizeA = numARows * numAColumns * sizeof(int);
    int sizeB = numBRows * numBColumns * sizeof(int);
    int sizeC = numCRows * numBColumns * sizeof(int);
    int sizeGold = numGoldRows * numGodlColumns * sizeof(int);

    hipError_t result = hipSuccess;

    //code
    //host memory allocation
    hostA = (int*)malloc(sizeA);
    if (hostA == NULL)
    {
        printf("Host Memory allocation is failed for hostA matrix.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    hostB = (int*)malloc(sizeB);
    if (hostB == NULL)
    {
        printf("Host Memory allocation is failed for hostB matrix.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    hostC = (int*)malloc(sizeC);
    if (hostC == NULL)
    {
        printf("Host Memory allocation is failed for hostC matrix.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    gold = (int*)malloc(sizeGold);
    if (gold == NULL)
    {
        printf("Host Memory allocation is failed for gold matrix.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    // printing matrix dimensions and sizes
    printf("The Dimensions of Matrix 'hostA' are : %d x %d\n",numARows,numAColumns);
    printf("The Dimensions of Matrix 'hostB' are : %d x %d\n",numBRows,numBColumns);
    printf("The Dimensions of Matrix 'hostC' are : %d x %d\n",numCRows,numCColumns);
    printf("The Dimensions of Matrix 'gold' are : %d x %d\n",numGoldRows,numGodlColumns);

    printf("Size of Matrix hostA = %d\n",sizeA);
    printf("Size of Matrix hostB = %d\n",sizeB);
    printf("Size of Matrix hostC = %d\n",sizeC);
    printf("Size of Matrix gold = %d\n",sizeGold);

    // fill source matrices
    InitA(hostA,numARows,numAColumns);
    InitB(hostB,numBRows,numBColumns);

    // device memory allocation
    result = hipMalloc((void**)&deviceA,sizeA);
    if (result != hipSuccess)
    {
        printf("Device Memory allocation is failed for deviceA matrix.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    result = hipMalloc((void**)&deviceB,sizeB);
    if (result != hipSuccess)
    {
        printf("Device Memory allocation is failed for deviceB array.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    result = hipMalloc((void**)&deviceC,sizeC);
    if (result != hipSuccess)
    {
        printf("Device Memory allocation is failed for deviceC array.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    //copy data from host matrices into device matrices
    result = hipMemcpy(deviceA, hostA, sizeA, hipMemcpyHostToDevice);
    if (result != hipSuccess)
    {
        printf("Host to Device Data Copy is failed for deviceA matrix.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    result = hipMemcpy(deviceB, hostB, sizeB, hipMemcpyHostToDevice);
    if (result != hipSuccess)
    {
        printf("Host to Device Data Copy is failed for deviceB array.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    // CUDA Kernel configuration
    dim3 dimGrid = dim3(ceil((int)numBColumns / (int)BLOCK_WIDTH), ceil((int)numARows / (int)BLOCK_WIDTH), 1);
    dim3 dimBlock = dim3(BLOCK_WIDTH, BLOCK_WIDTH, 1);

    // cuda kernel for Matrix Multiplication
    StopWatchInterface* timer = NULL;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    matMulGPU <<<dimGrid, dimBlock >>>(deviceA,deviceB,deviceC,numARows,numAColumns,numBColumns,numCColumns);

    sdkStopTimer(&timer);
    timeOnGPU = sdkGetTimerValue(&timer);
    timer = NULL;

    // copy data from Device to Host
    result = hipMemcpy(hostC, deviceC, sizeC, hipMemcpyDeviceToHost);
    if (result != hipSuccess)
    {
        printf("Device to Host Data copy is failed for deviceC array.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }
    
    // matrix multiplication on host
    matMulCPU(hostA, hostB, gold, numARows, numAColumns, numBColumns, numCColumns);

    // comparison
    int breakValue = -1;
    bool bAccuracy = true;
    for (int i = 0; i < numCRows * numCColumns; i++)
    {
        int val1 = gold[i];
        int val2 = hostC[i];
        if (val1 != val2)
        {
            breakValue = i;
            bAccuracy = false;
            break;
        }
    }

    char str[128];
    if(bAccuracy != true)
    {
        sprintf(str,"Comparison of CPU and GPU Matrix Multiplication is not accurate at array index %d",breakValue);
    }
    else
    {
        sprintf(str,"Comparison of CPU and GPU Matrix Multiplication is accurate");
    }

    printf("Time taken for Matrix Multiplication on CPU = %.6f\n",timeOnCPU);
    printf("Time taken for Matrix Multiplication on GPU = %.6f\n",timeOnGPU);
    printf("%s\n",str);

    // cleanup
    cleanup();

    return (0);
}

void InitA(int *data,int row,int col)
{
    int num = 1;

    //code
    for(int i = 0;i < row; i++)
    {
        for(int j = 0; j < col; j++)
        {
            *(data + i * row + j) = num;
            num++;
        }
    }
}

void InitB(int *data,int row,int col)
{
    int num = BLOCK_WIDTH;
    
    // code
    for (int i = 0; i < row; i++)
    {
        for(int j = 0; j < col; j++)
        {
            *(data + i * row + j) = num;
            num++;
        }
    }
}

void matMulCPU(int* A, int* B, int* C, int numARows, int numAColumns, int numBColumns, int numCColums)
{
    // code
    StopWatchInterface* timer = NULL;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);
        
    for (int i = 0; i < numARows; i++)
    {
        for (int j = 0; j < numBColumns; j++)
        {   
            int val = 0;
            for(int k = 0; k < numAColumns; k++)
            {
                int a = A[i * numAColumns + k];
                int b = B[k * numBColumns + j];
                val += a*b;
            }
            C[i* numCColums + j] = val;
        }
    }

    sdkStopTimer(&timer);
    timeOnCPU = sdkGetTimerValue(&timer);
    timer = NULL;
}

void cleanup(void)
{
    // code
    if(deviceC)
    {
        hipFree(deviceC);
        deviceC = NULL;
    }

    if(deviceB)
    {
        hipFree(deviceB);
        deviceB = NULL;
    }

    if(deviceA)
    {
        hipFree(deviceA);
        deviceA = NULL;
    }

    if(gold)
    {
        hipFree(gold);
        gold = NULL;
    }

    if(hostC)
    {
        hipFree(hostC);
        hostC = NULL;
    }

    if(hostB)
    {
        hipFree(hostB);
        hostB = NULL;
    }

    if(hostA)
    {
        hipFree(hostA);
        hostA = NULL;
    }

}
